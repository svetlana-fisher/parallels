#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <chrono>
#include <boost/program_options.hpp>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

namespace po = boost::program_options;

void init_matrix(double* matrix, int n) {
    matrix[n+1] = 10.0;
    matrix[n - 1 + n - 1] = 20.0;
    matrix[(n - 1) * (n - 1)] = 20.0;
    matrix[(n - 1) * (n - 1) + (n - 1) - 2] = 30.0;

    for (int j = 2; j < n - 2; j++) {
        matrix[n + j] = matrix[n + 1] + (matrix[n - 1 + n - 1] - matrix[n + 1]) * (static_cast<double>(j-1) / static_cast<double>(n - 3));
        matrix[(n - 1) * (n - 1) + j - 1] = matrix[(n - 1) * (n - 1)] + (matrix[(n - 1) * (n - 1) + (n - 1) - 2] - matrix[(n - 1) * (n - 1)]) * (static_cast<double>(j-1) / static_cast<double>(n - 3));
        matrix[j * n + 1] = matrix[n + 1] + (matrix[(n - 1) * (n - 1)] - matrix[n + 1]) * (static_cast<double>(j-1) / static_cast<double>(n - 3));
        matrix[j * n + (n - 1) - 1] = matrix[n - 1 + n - 1] + (matrix[(n - 1) * (n - 1) + (n - 1) - 2] - matrix[n - 1 + n - 1]) * (static_cast<double>(j-1) / static_cast<double>(n - 3));
    }
}

__global__ void jacobi_kernel(double* src, double* dst, int width) {
    int i = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int j = blockIdx.x * blockDim.x + threadIdx.x + 1;
    
    if (i < width - 1 && j < width - 1) {
        dst[i * width + j] = 0.25 * (src[i * width + j - 1] + 
                                    src[i * width + j + 1] + 
                                    src[(i - 1) * width + j] + 
                                    src[(i + 1) * width + j]);
    }
}

__global__ void diff_kernel(double* a, double* b, double* diff, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        diff[idx] = fabs(a[idx] - b[idx]);
    }
}

double compute_max_diff(double* d_a, double* d_b, int size) {
    double* d_diff;
    hipMalloc(&d_diff, size * sizeof(double));
    
    dim3 blockDim(256);
    dim3 gridDim((size + blockDim.x - 1) / blockDim.x);
    diff_kernel<<<gridDim, blockDim>>>(d_a, d_b, d_diff, size);
    hipDeviceSynchronize();
    
    void* d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    double max_diff;
    double* d_max_diff;
    hipMalloc(&d_max_diff, sizeof(double));
    
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_diff, d_max_diff, size);
    hipMalloc(&d_temp_storage, temp_storage_bytes);
    hipcub::DeviceReduce::Max(d_temp_storage, temp_storage_bytes, d_diff, d_max_diff, size);
    hipMemcpy(&max_diff, d_max_diff, sizeof(double), hipMemcpyDeviceToHost);
    
    hipFree(d_temp_storage);
    hipFree(d_max_diff);
    hipFree(d_diff);
    
    return max_diff;
}

int main(int argc, char** argv) {
    int n = 20;
    double accuracy = 1e-6;
    int max_iteration = 1000000;

    po::options_description desc("Allowed options");
    desc.add_options()
        ("help", "show help message")
        ("size", po::value<int>(&n), "grid size (128, 256, 512, 1024)")
        ("accuracy", po::value<double>(&accuracy), "desired accuracy")
        ("max_iter", po::value<int>(&max_iteration), "maximum number of iterations");

    po::variables_map vm;
    po::store(po::parse_command_line(argc, argv, desc), vm);
    po::notify(vm);

    if (vm.count("help")) {
        std::cout << desc << "\n";
        return 0;
    }

    const int width = n + 2;
    const size_t matrix_size = width * width * sizeof(double);

    double* h_matrix1 = new double[width * width];
    double* h_matrix2 = new double[width * width];
    
    init_matrix(h_matrix1, width);
    init_matrix(h_matrix2, width);

    double *d_matrix1, *d_matrix2;
    hipMalloc(&d_matrix1, matrix_size);
    hipMalloc(&d_matrix2, matrix_size);

    hipMemcpy(d_matrix1, h_matrix1, matrix_size, hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, h_matrix2, matrix_size, hipMemcpyHostToDevice);

    dim3 block_size(16, 16);
    dim3 grid_size((width + block_size.x - 1) / block_size.x, 
                   (width + block_size.y - 1) / block_size.y);

    int iter = 0;
    double error = 1.0;
    bool use_matrix1 = true;

    hipGraph_t graph;
    hipGraphExec_t graphExec;

    auto start = std::chrono::steady_clock::now();

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
    
    for (int i = 0; i < 500; i++) {
      if (use_matrix1) {
        jacobi_kernel<<<grid_size, block_size, 0, stream>>>(d_matrix1, d_matrix2, width);
      } else {
        jacobi_kernel<<<grid_size, block_size, 0, stream>>>(d_matrix2, d_matrix1, width);
      }
      use_matrix1 = !use_matrix1;
    }

    hipStreamEndCapture(stream, &graph);
    hipGraphInstantiate(&graphExec, graph, NULL, NULL, 0);
    hipStreamDestroy(stream);

    while (error > accuracy && iter < max_iteration) {
      hipGraphLaunch(graphExec, 0);
      hipDeviceSynchronize();
      
      iter += 500;
      
      error = compute_max_diff(use_matrix1 ? d_matrix1 : d_matrix2, 
                              use_matrix1 ? d_matrix2 : d_matrix1, 
                              width * width);
    }

    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    std::cout << "Iterations: " << iter << "\n";
    std::cout << "Final error: " << error << "\n";
    std::cout << "Time elapsed: " << elapsed.count() << " seconds" << std::endl;

    hipGraphExecDestroy(graphExec);
    hipGraphDestroy(graph);

    double* result_matrix = use_matrix1 ? h_matrix1 : h_matrix2;
    hipMemcpy(result_matrix, use_matrix1 ? d_matrix1 : d_matrix2, matrix_size, hipMemcpyDeviceToHost);

    std::ofstream out_file("result.dat", std::ios::binary);
    out_file.write(reinterpret_cast<const char*>(result_matrix), width * width * sizeof(double));
    out_file.close();

    delete[] h_matrix1;
    delete[] h_matrix2;
    hipFree(d_matrix1);
    hipFree(d_matrix2);

    return 0;
}